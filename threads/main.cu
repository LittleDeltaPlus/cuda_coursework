
#include <hip/hip_runtime.h>
#include <iostream>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <omp.h>
#include <sstream>
#include <fstream>

__global__ void gaussian (double delta, double a, double len, const double *x, double *y){
    unsigned int idx = blockIdx.x*blockDim.x +threadIdx.x;

    if (idx < len){
        //almost verbatim the summation formula
        y[idx] = delta*std::exp(-((a + delta*(x[idx]-0.5))*(a + delta*(x[idx]-0.5))));
    }
}


int main() {
    //remove existing benchmark file
    if(remove("gpuOut.txt") !=0){
        perror("error deleting file");
    }
    //Open Refrence Benchmark file
    std::ofstream outputFile ("gpuOut.txt");
    std::ifstream infile ("../cpuOut_omp.txt");
    //Initialise Numerical Integration Variables
    const double b = 10000000,
            a = -b;
    const int  nUpLim = (int)b*3, nLoLim =(int)(b*0.1);// (int)b;


    for(int n = nLoLim; n <= nUpLim; n+=nLoLim){
        //load the corresponding benchmark
        std::string line;
        getline(infile, line);
        std::istringstream iss(line);
        long double delta_f, sum_lf, duration_lf, sum_of, duration_of, speedup_of;
        if(!(iss >> delta_f >> sum_lf >> duration_lf >> sum_of >> duration_of >> speedup_of)){std::cout << "oops"<<std::endl;}
        //ensure device is ready
        hipDeviceSynchronize();
        //initialise Device vairables
        double *d_x, *d_y;
        hipMalloc(&d_x, n*sizeof(double));
        hipMalloc(&d_y, n*sizeof(double));
        //initialise Host variables
        auto *x = new double[n], *y = new double[n];

        //start timer
        long double t1 = omp_get_wtime();
        //calculate delta
        double delta = (b-a)/n;

        //populate host variables
        for (int i=0; i<=n ; i++){
            x[i]=i;
            y[i]=0;
        }
        //copy host variables to the device
        hipMemcpy(d_x, x, n*sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_y, y, n*sizeof(double), hipMemcpyHostToDevice);
        //evaluate the gaussian function
        gaussian<<<n/1024,1024>>>(delta, a, n, d_x, d_y);
        //copy the result back
        hipMemcpy(y, d_y, n*sizeof(double), hipMemcpyDeviceToHost);
        //Sum the output
        double thread_sum=0;
        for (int i=0; i<=n ; i++){
            thread_sum+=y[i];
        }
        //end the timer
        long double t2 = omp_get_wtime();
        long double thread_duration = (t2 - t1 );
        //write the benchmark for this delta
        outputFile << std::setprecision(18) << delta_f << " " << sum_lf << " " << duration_lf << " " << sum_of << " " << duration_of << " " << speedup_of
                   << " " << thread_sum << " " << thread_duration << " " << duration_lf / thread_duration << " " << duration_of / thread_duration << std::endl;
        //indicate this pass is done
        std::cout << n << std::endl;

    }
    //close the benchmark output and finish
    outputFile.close();
    return 0;
}
